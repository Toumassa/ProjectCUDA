#include "hip/hip_runtime.h"
#include "kernel.h"



void copyTreeToGPU(ANode *cpuTree, ANode**gpuTree, int treeSize)
{
	hipError_t ok;
	size_t size;
	
	// Allocate GPU memory for the features and transfer
	// them from host memory to GPU memory
	size=treeSize*sizeof(ANode);
	ok=hipMalloc ((void**) gpuTree, size);
	
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu allocation for tree:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	//CHECK_CUDA_MALLOC;
	ok=hipMemcpy (*gpuTree, cpuTree, size, hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cerr << "Error memcpy RAM to GPU for tree storage\n";
		exit(1);
	}
	
	
}


void copyFeaturesToGPU(float *features, int fsize, float *integral_features, int fintegral_size, float **_features, float **_integral_features)
{
	hipError_t ok;
	size_t size;
	
	size = fsize*sizeof(float);
	ok=hipMalloc ((void**) _features, size);
	
	
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu allocation for features:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	ok=hipMemcpy (*_features, features, size, hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu memcpy for features:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	
	size = fintegral_size*sizeof(float);
	ok=hipMalloc ((void**) _integral_features, size);
	
	
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu allocation for features_integral:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	ok=hipMemcpy (*_integral_features, integral_features, size, hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu memcpy for features_integral:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
}


void copyCommonHistTabToGPU(uint32_t*hist, uint32_t**_hist, int hsize)
{
	hipError_t ok;
	size_t size;
	
	size = hsize*sizeof(uint32_t);
	ok=hipMalloc ((void**) _hist, size);
	
	
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu allocation for features:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	ok=hipMemcpy (*_hist, hist, size, hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu memcpy for features:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	
	ok=hipMemcpy (hist,*_hist, size, hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpu memcpy for features:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	/*for(int i = 0; i < hsize; i++)
	cout << "*h=" << hist[i] <<endl;*/
}

void GPUAdapter::PushTreeToGPU(int n)
{
	if(n < 0 || n > this->treesAsVector.size())
	{
		cerr << "PushTreeToGPU: incorrect tree index" << endl;
		exit(1);
	}

	//change with malloc GPU
	ANode *treeToGPU;
	
	/*for(int i = 0; i < this->treesAsVector[n]->size();i++)
		tree[i]=(*this->treesAsVector[n])[i];*/
	
	copyTreeToGPU(this->treesAsVector[n]->data(), &treeToGPU, this->treesAsVector[n]->size());
	
	_treeAsTab.push_back(treeToGPU);
}

__device__
float gpuGetValue (float *gpuFeatures, uint8_t channel, 
    int16_t x, int16_t y, int16_t w, int16_t h)
{
  //cout << "before gpuGetValue\n";
    float res = gpuFeatures[y+x*h + channel*w*h];
    return res;
}

__device__
float gpuGetValueIntegral (float *gpuFeaturesIntegral, uint8_t channel, 
    int16_t x1, int16_t y1, int16_t x2, int16_t y2, int16_t w, int16_t h)
{
    float res = (
            gpuFeaturesIntegral[y2 + x2*h + channel*w*h] -
            gpuFeaturesIntegral[y2 + x1*h + channel*w*h] -
            gpuFeaturesIntegral[y1 + x2*h + channel*w*h] +
            gpuFeaturesIntegral[y1 + x1*h + channel*w*h]);

    return res;
}


__device__
SplitResult split(SplitData<float> splitData, Sample<float> &sample, int16_t w, int16_t h, int16_t w_i, int16_t h_i, float *gpuFeatures, float *gpuFeaturesIntegral)
{
   sample.value = gpuGetValue(gpuFeatures, splitData.channel0, sample.x, sample.y, w, h);
	SplitResult centerResult = (sample.value < splitData.thres) ? SR_LEFT : SR_RIGHT;
    if (splitData.fType == 0) // single probe (center only)
    {
        return centerResult;
    }
    // for cases when we have non-centered probe types
    int pt1x, pt1y, pt2x, pt2y, pt3x, pt3y, pt4x, pt4y;

    pt1x = sample.x + splitData.dx1 - splitData.bw1;
    pt1y = sample.y + splitData.dy1 - splitData.bh1;

    pt2x = sample.x + splitData.dx1 + splitData.bw1 + 1; // remember -> integral images have size w+1 x h+1
    pt2y = sample.y + splitData.dy1 + splitData.bh1 + 1;

    if (pt1x < 0 || pt2x < 0 || pt1y < 0 || pt2y < 0 ||
        pt1x > w || pt2x > w || pt1y > h || pt2y > h) // due to size correction in getImgXXX we dont have to check \geq
    {
      return centerResult;
    }
    else
    {
      if (splitData.fType == 1) // single probe (center - offset)
      {
        int16_t norm1 = (pt2x - pt1x) * (pt2y - pt1y);
        sample.value -= gpuGetValueIntegral(gpuFeaturesIntegral, splitData.channel0, pt1x, pt1y, pt2x, pt2y, w_i, h_i) / norm1;
      }
      else                      // pixel pair probe test
      {
        pt3x = sample.x + splitData.dx2 - splitData.bw2;
        pt3y = sample.y + splitData.dy2 - splitData.bh2;

        pt4x = sample.x + splitData.dx2 + splitData.bw2 + 1;
        pt4y = sample.y + splitData.dy2 + splitData.bh2 + 1;


        if (pt3x < 0 || pt4x < 0 || pt3y < 0 || pt4y < 0 ||
            pt3x > w || pt4x > w || pt3y > h || pt4y > h)
        {
          return centerResult;
        }

        int16_t norm1 = (pt2x - pt1x) * (pt2y - pt1y);
        int16_t norm2 = (pt4x - pt3x) * (pt4y - pt3y);

        if (splitData.fType == 2)    // sum of pair probes
        {
          sample.value = gpuGetValueIntegral(gpuFeaturesIntegral, splitData.channel0, pt1x, pt1y, pt2x, pt2y, w_i, h_i) / norm1
                       + gpuGetValueIntegral(gpuFeaturesIntegral, splitData.channel1, pt3x, pt3y, pt4x, pt4y, w_i, h_i) / norm2;
        }
        else if (splitData.fType == 3)  // difference of pair probes
        {
          sample.value = gpuGetValueIntegral(gpuFeaturesIntegral, splitData.channel0, pt1x, pt1y, pt2x, pt2y, w_i, h_i) / norm1
                       - gpuGetValueIntegral(gpuFeaturesIntegral, splitData.channel1, pt3x, pt3y, pt4x, pt4y, w_i, h_i) / norm2;
        }

      }
    }
    SplitResult res = (sample.value < splitData.thres) ? SR_LEFT : SR_RIGHT;

    return res;
}
__device__
void predict(int *returnStartHistTab, ANode* tree, int16_t w, int16_t h, int16_t w_i, int16_t h_i, float* features, float* features_integral, Sample<float> sample)
{
  int curNode = 0; //initialising to Root
    SplitResult sr = SR_LEFT;
    while (tree[curNode].left != -1 && sr != SR_INVALID)
    {
    sr = split(tree[curNode].splitData, sample, w, h, w_i, h_i, features, features_integral);
   
    switch (sr)
      {
      case SR_LEFT:
        curNode = tree[curNode].left;
        break;
      case SR_RIGHT:
        curNode = tree[curNode].right;
        break;
      default:
        break;
      }
    }
    (*returnStartHistTab) = tree[curNode].common_hist_tab_offset;
}
/**
	Copies data which doesnt depend on images
*/
void GPUAdapter::init(StrucClassSSF<float> *forest, ConfigReader *cr)
{
	this->treeTabCount = cr->numTrees;
	for(size_t t = 0; t < this->treeTabCount; ++t)
    {
    	this->AddTree(&(forest[t]));
    }

	for(int i = 0; i < this->treeTabCount; i++)
    {
    	PushTreeToGPU(i);
    }
	
	copyCommonHistTabToGPU(common_hist_tab.data(), &_common_hist_tab, this->common_hist_tab.size());
	
	common_hist_tab.clear();
}
void GPUAdapter::preKernel(uint16_t imageId, ConfigReader *cr, TrainingSetSelection<float> *pTS)
{
	hipError_t ok;
    this->ts = pTS;
    
    this->pImageData = this->ts->pImageData;
    this->nChannels = this->ts->getNChannels();
    this->iWidth = this->ts->getImgWidth(0);
    this->iHeight = this->ts->getImgHeight(0);
    this->numLabels = cr->numLabels;
    this->lPXOff = cr->labelPatchWidth / 2;
    this->lPYOff = cr->labelPatchHeight / 2;

	
  
    
    this->getFlattenedFeatures(imageId, &(this->features), &(this->nChannels));
    this->getFlattenedIntegralFeatures(imageId, &(this->features_integral), &(this->w_integral), &(this->h_integral));

    copyFeaturesToGPU(this->features, this->fSize, 
						this->features_integral, this->fIntegralSize, 
						&this->_features, &this->_features_integral);
	
	
	int size = this->iWidth*this->iHeight*this->numLabels*sizeof(int);
	this->result = (int*)malloc(size);
	
	/*for(int i =0; i < this->iWidth*this->iHeight*this->numLabels; i++)
	{
		this->result[i] = 0;
	}*/
    
	ok = hipMalloc((void**) &this->resultGPU, size);if(ok != hipSuccess)
	{
		std::cerr << "Error gpu allocation this->resultGPU:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	ok= hipMemset  ( this->resultGPU,
                            0,
                           size    
                        )  ;
	if(ok != hipSuccess)
	{
		std::cerr << "Error gpuresult initializing:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
	ok=hipMemcpy (this->resultGPU, this->result, size, hipMemcpyHostToDevice);
	if(ok != hipSuccess)
	{
		std::cerr << "Error memcpy RAM to GPU for tree this->result\n";
		exit(1);
	}
	if(this->result == NULL)
	{
		std::cerr << " HOST memory allocation failed" << endl;
		exit(1);
	}
	delete [] this->features;
	delete [] this->features_integral;
}

__global__
void kernel(int *result, ANode* tree, int16_t w, int16_t h, int16_t w_i, int16_t h_i, float* features, float* features_integral, 
		Sample<float> sample, int lPXOff, int lPYOff, uint32_t *common_hist_tab, int numLabels)
{
	int sx = blockIdx.x*blockDim.x+threadIdx.x;
	int sy = blockIdx.y*blockDim.y+threadIdx.y;
	
	int p;
	sample.x = sx;
	sample.y = sy;
	predict(&p, tree, w, h, w_i, h_i, features, features_integral, sample);
	
	int ptx, pty;
	for (pty=(int)sy-lPYOff;pty<=(int)sy+(int)lPYOff;++pty)
	for (ptx=(int)sx-(int)lPXOff;ptx<=(int)sx+(int)lPXOff;++ptx,++p)
	{
		if (common_hist_tab[p]< 0 || common_hist_tab[p] >= numLabels)
		{
			/*cout << "x:" << sx << " y:"<<sy << " tree:"<< t << endl;
			cout << "pt.x:" << pt.x << " pt.y:"<<pt.y << ":"<< p << endl;
			cout << "*p : " << common_hist_tab[p] << endl;
			//std::cerr << "Invalid label in prediction: " << (int) common_hist_tab[p] << "\n";
			*///exit(1);
		}         
		else if (ptx >=0 && ptx<w && pty >= 0 && pty < h)
		{	
			result[common_hist_tab[p]*w*h+w*pty+ptx]+=1;
		}
	}
}

void GPUAdapter::testGPUSolution(cv::Rect box, Sample<float>&s)
{
	int blockSize = 32;

	s.x = 0;
	s.y = 0;

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(box.width/blockSize, box.height/blockSize);
    
    for(size_t t = 0; t < this->_treeAsTab.size(); ++t)
    {
		
		kernel<<<dimGrid, dimBlock>>>
		(this->resultGPU, _treeAsTab[t], 
		this->iWidth, this->iHeight, 
		this->w_integral, this->h_integral, 
		this->_features, this->_features_integral, 
		s, 
		this->lPXOff, this->lPYOff, 
		this->_common_hist_tab,  
		this->numLabels
		);
	}
	
}

void GPUAdapter::postKernel(cv::Mat*mapResult)
{
	hipError_t ok;
	int size = this->iWidth*this->iHeight*this->numLabels*sizeof(int);
	ok=hipMemcpy (this->result, this->resultGPU, size, hipMemcpyDeviceToHost);
	if(ok != hipSuccess)
	{
		std::cerr << "Cant get this->result back:"<<hipGetErrorString(ok)<<"\n";
		exit(1);
	}
    int ptx, pty;
    size_t maxIdx;
    cv::Point pt;
    for (pty = 0; pty < this->iHeight; ++pty)
    for (ptx = 0; ptx < this->iWidth; ++ptx)
    {
        maxIdx = 0;

	
        for(int j = 1; j < this->numLabels; ++j)
        {
			if(this->result[j*this->iWidth*this->iHeight+pty*this->iWidth+ptx] > this->result[maxIdx*this->iWidth*this->iHeight+pty*this->iWidth+ptx])
				maxIdx = j;
            
			
        }
		pt.x = ptx;
		pt.y = pty;
        (*mapResult).at<uint8_t>(pt) = (uint8_t)maxIdx;
    }
	
	delete [] this->result;
	hipFree(this->resultGPU);
	hipFree(this->_features);
	hipFree(this->_features_integral);
}


void GPUAdapter::destroy()
{
	for(int i=0; i < _treeAsTab.size();i++)
	{
		hipFree(_treeAsTab[i]);
	}
	hipFree(_common_hist_tab);
}
